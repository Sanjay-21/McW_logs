#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>

const int N = 1000000;

// CUDA kernel to add two vectors
__global__ void addVectors(int *a, int *b, int *c, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    // Host vectors
    std::vector<int> h_a(N, 1);
    std::vector<int> h_b(N, 2);
    std::vector<int> h_c(N);

    // Device vectors
    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, N * sizeof(int));
    hipMalloc((void **)&d_b, N * sizeof(int));
    hipMalloc((void **)&d_c, N * sizeof(int));

    // Copy host vectors to device
    hipMemcpy(d_a, h_a.data(), N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), N * sizeof(int), hipMemcpyHostToDevice);

    // Set up CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Define block and grid sizes
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    // Record the start event
    hipEventRecord(start);

    // Launch the CUDA kernel
    addVectors<<<gridSize, blockSize>>>(d_a, d_b, d_c, N);

    // Record the stop event
    hipEventRecord(stop);

    // Synchronize to make sure the kernel is done
    hipEventSynchronize(stop);

    // Copy the result back to the host
    hipMemcpy(h_c.data(), d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // Calculate the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << milliseconds << "\n";

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
