#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>

const int N = 1000000;

// CUDA kernel to add two vectors
__global__ void addVectors(int *a, int *b, int *c, int n, int *temp) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    temp[tid]=threadIdx.x;
    if (tid < n) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    // Host vectors
    std::vector<int> h_a(N, 1);
    std::vector<int> h_b(N, 2);
    std::vector<int> h_c(N);
    std::vector<int> temp(N);

    // Device vectors
    int *d_a, *d_b, *d_c, *temp1;
    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_b, N * sizeof(int));
    hipMalloc(&d_c, N * sizeof(int));
    hipMalloc(&temp1, N * sizeof(int));

    // Copy host vectors to device
    hipMemcpy(d_a, h_a.data(), N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), N * sizeof(int), hipMemcpyHostToDevice);
    //cudaMemcpy(temp1, temp.data(), N * sizeof(int), cudaMemcpyHostToDevice);
    std::cout<<" Size: "<<sizeof(d_a)<<" N: "<<sizeof(h_a)<<"\n";
    // Set up CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Define block and grid sizes
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    // Record the start event
    hipEventRecord(start);

    // Launch the CUDA kernel
    addVectors<<<gridSize, blockSize>>>(d_a, d_b, d_c, N, temp1);

    // Record the stop event
    hipEventRecord(stop);

    // Synchronize to make sure the kernel is done
    hipEventSynchronize(stop);

    // Copy the result back to the host
    hipMemcpy(h_c.data(), d_c, N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(temp.data(), temp1, N * sizeof(int), hipMemcpyDeviceToHost);
    std::cout<<temp[1]<<"\n";

    // Calculate the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << milliseconds << "\n";

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
