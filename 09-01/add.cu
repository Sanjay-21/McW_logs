
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#define N 999

__global__ void add_vectors(double *a, double *b, double *c)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if(id < N) c[id] = a[id] + b[id];
}


int main()
{
    using std::chrono::high_resolution_clock;
    using std::chrono::duration_cast;
    using std::chrono::duration;
    using std::chrono::milliseconds;

	size_t bytes = N*sizeof(double);

	double *A = (double*)malloc(bytes);
	double *B = (double*)malloc(bytes);
	double *C = (double*)malloc(bytes);

	double *d_A, *d_B, *d_C;
	hipMalloc(&d_A, bytes);
	hipMalloc(&d_B, bytes);
	hipMalloc(&d_C, bytes);

	for(int i=0; i<N; i++)
	{
		A[i] = 1.0;
		B[i] = 2.0;
	}

    auto t1 = high_resolution_clock::now();

	hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice);

	int thr_per_blk = 256;
	int blk_in_grid = ceil(float(N) / thr_per_blk);


	add_vectors<<< blk_in_grid, thr_per_blk >>>(d_A, d_B, d_C);


	hipMemcpy(C, d_C, bytes, hipMemcpyDeviceToHost);

    auto t2 = high_resolution_clock::now();
    duration<double, std::milli> ms_double = t2 - t1;
    std::cout << "CUDA: "<<ms_double.count() << "ms\n";



    t1 = high_resolution_clock::now();
    for(int i=0; i<N; i++)
	{
		C[i] = A[i]+B[i];
	}
    t2 = high_resolution_clock::now();
    ms_double = t2 - t1;
    std::cout << "C++: "<<ms_double.count() << "ms\n";

    for(int i=0,j=0;i<N;i++)
    {
        if(j<9){
            std::cout<<C[i]<<"\t";
            j++;
        }
        else{
            std::cout<<C[i]<<"\n";
            j=0;
        }
            
    }
    
}